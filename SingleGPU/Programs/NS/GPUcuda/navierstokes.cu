#include "hip/hip_runtime.h"
/*--------------------------------------------------------------------
*
* PURPOSE
*
* This program numerically solves the 2D incompressible Navier-Stokes
* on a Square Domain [0,1]x[0,1] using pseudo-spectral methods and
* Crank-Nicolson timestepping. The numerical solution is compared to
* the exact Taylor-Green Vortex Solution. 
*
* Periodic free-slip boundary conditions and Initial conditions:
*	u(x,y,0)=sin(2*pi*x)cos(2*pi*y)
*	v(x,y,0)=-cos(2*pi*x)sin(2*pi*y)
* Analytical Solution (subscript denote derivatives):
*	u(x,y,t)=sin(2*pi*x)cos(2*pi*y)exp(-8*pi^2*t/Re)
*	v(x,y,t)=-cos(2*pi*x)sin(2*pi*y)exp(-8*pi^2*t/Re)
*   u_y(x,y,t)=-2*pi*sin(2*pi*x)sin(2*pi*y)exp(-8*pi^2*t/Re)
*	v_x(x,y,t)=2*pi*sin(2*pi*x)sin(2*pi*y)exp(-8*pi^2*t/Re)
*	omega=v_x-u_y
*
* AUTHORS
*
* B. Cloutier, B.K. Muite, P. Rigge
* 4 June 2012
*
* .. Scalars ..
*  Nx				= number of modes in x - power of 2 for FFT
*  Ny				= number of modes in y - power of 2 for FFT
*  nplots			= number of plots produced
*  plotgap			= number of timesteps inbetween plots
*  Re 				= Reynold's number
*  dt				= timestep size 
*  tol				= determines when convergences is reached
*  i				= loop counter in x direction
*  j				= loop counter in y direction
*  n				= loop counter for timesteps between plots	
*  nn 				= loop counter for plots
*  chg				= error at each iteration	
*  max				= maximum error
*  pi				= value of pi
*  xsize    		= size of real arrays in x direction
*  ysize    		= size of real arrays in y direction
*  gridsize 		= size of array for complex data
*  start_time		= start time of computation
*  end_time			= end time of evaluation
*  pland2z			= Forward 2d fft plan  (CUFFT)
*  planz2d 			= Backward 2d fft plan (CUFFT)
*  nThreads			= Number of threads for GPU to use
*  nBlocksR			= number of blocks for GPU to use for real arrays
*  nBlocksC			= number of blocks for GPU to use for complex arrays
*
* .. Arrays on CPU ..
*
*  u				= velocity in x direction
*  v				= velocity in y direction
*  omeg				= vorticity	in real space
*  omegold			= vorticity in real space at previous
*						iterate
*  omegexact		= taylor-green vorticity at
*						at final step
*  x				= x locations
*  y				= y locations
*
* .. Arrays on GPU ..
*
*  u_d				= velocity in x direction
*  v_d				= velocity in y direction
*  omeg_d			= vorticity	in real space
*  omegold			= vorticity in real space at previous
*						iterate
*  omegoldhat_d		= 2D Fourier transform of vorticity at previous
*						iterate
*  nlhat_d			= nonlinear term in Fourier space
*  nloldhat_d		= nonlinear term in Fourier space
*						at previous iterate
*  omegexact		= taylor-green vorticity at
*						at final step
*  psihat_d			= 2D Fourier transform of streamfunction
*						at next iteration
*  omegcheck_d		= store of vorticity at previous iterate
*  temp1_d 			= temporary real space used for
*						calculations.
*  temp2_d 			= temporary complex space used for
*						calculations. 
*  kx_d				= fourier frequencies in x direction
*  ky_d				= fourier frequencies in y direction
*  x_d				= x locations
*  y_d				= y locations
*
* REFERENCES
*
* ACKNOWLEDGEMENTS
*
* The format for the complex data types and style has followed examples on 
* the Nvidia website
*
* ACCURACY
*		
* ERROR INDICATORS AND WARNINGS
*
* FURTHER COMMENTS
* Check that the initial iterate is consistent with the
* boundary conditions for the domain specified
*--------------------------------------------------------------------
* External libraries required
*       Cuda FFT
*/


#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>

#include <hipfft/hipfft.h>
#include <cutil_inline.h>

typedef double2 cmplx;
typedef double real;
typedef hipfftDoubleReal cufftReal_;
typedef hipfftDoubleComplex cufftCmplx;
#define cufftMyExecF hipfftExecD2Z
#define cufftMyExecB hipfftExecZ2D
#define CUFFT_TFORM_FORWARD  HIPFFT_D2Z
#define CUFFT_TFORM_BACKWARD HIPFFT_Z2D

static __device__ __host__ inline cmplx cmul(cmplx a, cmplx b){
	cmplx c;
	c.x = a.x * b.x - a.y * b.y;
	c.y = a.x * b.y + a.y * b.x;
	return c;
}

static __device__ inline cmplx cadd (cmplx a, cmplx b) {
  cmplx c;
  c.x = a.x + b.x;
  c.y = a.y + b.y;
  return c;
}

static __device__ inline cmplx csub (cmplx a, cmplx b) {
  cmplx c;
  c.x = a.x - b.x;
  c.y = a.y - b.y;
  return c;
}

static __device__ inline cmplx cscale(cmplx a, real b) {
	cmplx c;
	c.x = a.x * b;
	c.y = a.y * b;
	return c;
}

void initialdata (int Nx, int Ny, real pi, real *x, real *y, real *omeg) {
	int i, j;
	for(j=0; j<Ny; j++){
		for(i=0; i<Nx; i++){
			omeg[Nx*j + i] = 4.0*pi*sin(2.0*pi*x[i])*sin(2.0*pi*y[j]);
		}
	}
}

static __global__ void nonlin1(int Ny, real *kx_d, cmplx *omeghat_d, cmplx *temp1_d) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int i;
  i = idx / (Ny/2+1);	
  cmplx c;
  c.x=(real)0.0;
  c.y=kx_d[i];
  temp1_d[idx]=cmul(omeghat_d[idx], c);
}

static __global__ void nonlin2(real *u_d, real *temp2_d, real *nl_d) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	nl_d[idx]=u_d[idx]*temp2_d[idx];
}

static __global__ void nonlin3(int Ny, real *ky_d, cmplx *omeghat_d, cmplx *temp1_d) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int j;
	cmplx c;
	j = idx % (Ny/2+1);	
	c.x=(real)0.0;
	c.y=ky_d[j];
	temp1_d[idx]=cmul(omeghat_d[idx], c);
}

static __global__ void nonlin4(int Nx, int Ny, real *nl_d, real *v_d, real *temp2_d) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	real scale= (real) 1./(real)(Nx*Ny);
	nl_d[idx]=(nl_d[idx]+v_d[idx]*temp2_d[idx])*scale;
}

static __global__ void nextstep1(int Nx, int Ny, real dt, real Re, real *kx_d, real *ky_d, cmplx *omeghat_d, cmplx *omegoldhat_d, cmplx *nloldhat_d, cmplx *nlhat_d) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int i,j;
	i = idx / (Ny/2+1);
	j = idx % (Ny/2+1);	
	
	real coef1 = (real) 1.*(-kx_d[i]*kx_d[i] - ky_d[j]*ky_d[j]);
	real coef2 = (real) 1./(dt);
	real coef3 = (real) 1./(Re);
	real coef4 = (real) 1.*(coef2 + (real)0.5*coef3*coef1);
	
	cmplx term1 = cscale (omegoldhat_d[idx],coef4);
	cmplx term2 = cadd (nloldhat_d[idx],nlhat_d[idx]);
	cmplx term3 = cscale (term2, (real)0.5);
	real coef5 = (real) 1./(coef2 - (real)0.5*coef3*coef1);
	omeghat_d[idx] = cscale(csub (term1,term3), coef5);

}

static __global__ void nextstep2(int Nx, int Ny, real *kx_d, real *ky_d, cmplx *omeghat_d, cmplx *psihat_d) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int i,j;
	i = idx / (Ny/2+1);
	j = idx % (Ny/2+1);	

	real coef6 = (real) -1./(-kx_d[i]*kx_d[i] - ky_d[j]*ky_d[j] + pow((real)0.10,14));
	psihat_d[idx] = cscale(omeghat_d[idx],coef6);	
}

static __global__ void nextstep3(int Nx, int Ny, cmplx *psihat_d, real *kx_d, cmplx *temp1_d) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int i;
	cmplx c;
	i = idx / (Ny/2+1);	
	c.x=(real)0.0;
	c.y=-kx_d[i]/(real)(Nx*Ny);
	temp1_d[idx]= cmul(c,psihat_d[idx]);
}
	
static __global__ void nextstep4(int Nx, int Ny, cmplx *psihat_d, real *ky_d, cmplx *temp1_d){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int j;
	cmplx c;
	j = idx % (Ny/2+1);
	c.x=(real)0.0;
	c.y=ky_d[j]/(real)(Nx*Ny);
	temp1_d[idx]=cmul(c,psihat_d[idx]);
}
			
static __global__ void copyRealArray(real *lhs, real *rhs) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	lhs[idx]=rhs[idx];
}

static __global__ void copyCmplxArray(cmplx *lhs, cmplx *rhs) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	lhs[idx].x=rhs[idx].x;
	lhs[idx].y=rhs[idx].y;
}

static __global__ void checkConvergence1(int Nx, int Ny, real *omeg_d, real *omegcheck_d) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	real scale=(real)1.0/(real)(Nx*Ny);
	omegcheck_d[idx] = ((omeg_d[idx]-omegcheck_d[idx])*scale);
	omegcheck_d[idx] = omegcheck_d[idx]*omegcheck_d[idx]; 
}


void savedata (int Nx, int Ny, int nplot, real *omeg) {
  FILE *f = NULL;
  char nameconfig1[128];
  nameconfig1[0]='\0';
  sprintf (nameconfig1, "data/omeg%.10d.datbin", nplot);
  f = fopen (nameconfig1, "wb");
  real *omegscale;
  omegscale = (real*)malloc (Nx * Ny * sizeof(real));
  int i;
  for (i=0; i<Nx*Ny; i++) {
    omegscale[i] = omeg[i]*1./(real)(Nx*Ny);
  }
  fwrite (omegscale, sizeof(real), Nx*Ny, f);
  fclose (f);
}

int main(){
	// declare variables
	hipfftHandle planz2d, pland2z;	
	real chg, max;
	real pi;
	int Nx=1024;
	int Ny=1024;
	int nplots=1;
	int plotgap=20;
	real Re = 1.e0;
	real tol = 10.e-10;
	real dt = 0.000125;
	int nThreads;
	int nBlocksR;
	int nBlocksC;
	real *kx,*ky;
	real *x, *y, *omeg, *omegexact;
	int i, j, n, nn;
	struct timeval start_time, end_time;
	// declare variables for GPU
	real *u_d, *v_d;
	real *omegcheck_d, *omeg_d, *nl_d, *temp2_d;
	cmplx *omegoldhat_d, *nloldhat_d, *omeghat_d, *nlhat_d, *psihat_d, *temp1_d;
	real *kx_d, *ky_d;
	real *x_d, *y_d;
	size_t xsize	= Nx * sizeof(real);
	size_t ysize	= Ny * sizeof(real);
	size_t gridsize = Nx * (Ny/2+1)*sizeof(cmplx);

	pi = 3.14159265358979323846264338327950288419716939937510;
	nThreads= (min(512, Nx));
	nBlocksR= (Nx * Ny / nThreads);
	nBlocksC= (Nx * (Ny/2+1) / nThreads);
	
	printf("Program starting\n");
	printf("Grid: %d X %d\n",Nx,Ny);
	printf("dt: %lf\n",dt);
	
	kx=(real*) malloc(xsize);
	ky=(real*) malloc(ysize);
	x=(real*) malloc(xsize);
	y=(real*) malloc(ysize);
	omeg=(real*) malloc(Nx * Ny * sizeof(real));
	omegexact=(real*) malloc(Nx * Ny * sizeof(real));

	printf("Allocated CPU arrays\n");
	
	cutilSafeCall(hipMalloc((void**)&u_d, Nx * Ny * sizeof(cufftReal_)));
	cutilSafeCall(hipMalloc((void**)&v_d, Nx * Ny * sizeof(cufftReal_)));
	cutilSafeCall(hipMalloc((void**)&omegcheck_d, Nx * Ny * sizeof(cufftReal_)));
	cutilSafeCall(hipMalloc((void**)&omeg_d, Nx * Ny * sizeof(cufftReal_)));
	cutilSafeCall(hipMalloc((void**)&nl_d, Nx * Ny * sizeof(cufftReal_)));
	cutilSafeCall(hipMalloc((void**)&temp2_d, Nx * Ny * sizeof(cufftReal_)));
	cutilSafeCall(hipMalloc((void**)&omegoldhat_d, gridsize));
	cutilSafeCall(hipMalloc((void**)&nloldhat_d, gridsize));
	cutilSafeCall(hipMalloc((void**)&omeghat_d, gridsize));
	cutilSafeCall(hipMalloc((void**)&nlhat_d, gridsize));
	cutilSafeCall(hipMalloc((void**)&psihat_d, gridsize));
	cutilSafeCall(hipMalloc((void**)&temp1_d, gridsize));
	cutilSafeCall(hipMalloc((void**)&kx_d, Nx * sizeof(cufftReal_)));
	cutilSafeCall(hipMalloc((void**)&ky_d, Ny * sizeof(cufftReal_)));
	cutilSafeCall(hipMalloc((void**)&x_d, Nx * sizeof(cufftReal_)));
	cutilSafeCall(hipMalloc((void**)&y_d, Ny * sizeof(cufftReal_)));
	printf("Allocated GPU arrays\n");

	cufftSafeCall(hipfftPlan2d(&pland2z, Nx, Ny, CUFFT_TFORM_FORWARD));
	cufftSafeCall(hipfftPlan2d(&planz2d, Nx, Ny, CUFFT_TFORM_BACKWARD));
	printf("Setup FFTs\n");
	
	// setup fourier frequencies
	for(i=0; i<Nx/2; i++)
		kx[i]=2.0*pi*(real)i; 			
	kx[Nx/2]=0;	
	for(i=0; i<Nx/2-1; i++)
		kx[Nx/2+1+i] = -kx[Nx/2-1-i];
	for(i=0; i<Nx; i++)
		x[i]=(real)i/(real)Nx; 
		
	for(j=0; j<Ny/2; j++)
		ky[j]=2.0*pi*(real)j;  			
	ky[Ny/2]=0.0;
	for(j=0; j<Ny/2-1; j++)
		ky[Ny/2+1+j]=-ky[Ny/2-1-j];
	for(j=0; j<Ny; j++)
		y[j]=(real)j/(real)Ny; 
		
	cutilSafeCall(hipMemcpy(kx_d,kx,Nx*sizeof(real),hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy(x_d,x,Nx*sizeof(real),hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy(ky_d,ky,Ny*sizeof(real),hipMemcpyHostToDevice));
	cutilSafeCall(hipMemcpy(y_d,y,Ny*sizeof(real),hipMemcpyHostToDevice));
	
	printf("Setup grid and fourier frequencies\n");

	//!!!!!!!!!!!!!!
	//!initial data!
	//!!!!!!!!!!!!!!
	initialdata(Nx,Ny,pi,x,y,omeg);
	cutilSafeCall(hipMemcpy(omeg_d,omeg,Ny*Nx*sizeof(real),hipMemcpyHostToDevice));
	printf("Copied initial data to device\n");
	
	copyRealArray <<< nBlocksR, nThreads >>>(omegcheck_d, omeg_d);
	cutilCheckMsg("Kernel execution failed: [ copyRealArray ]");
	cufftSafeCall(cufftMyExecF(pland2z,(cufftReal_ *)omeg_d,(cufftCmplx *)omeghat_d));

	nextstep2 <<< nBlocksC, nThreads >>> (Nx, Ny, kx_d, ky_d, omeghat_d, psihat_d);
	cutilCheckMsg("Kernel execution failed: [ nextstep2 ]");
 				
	nextstep3 <<< nBlocksC, nThreads >>>(Nx,Ny,psihat_d,kx_d,temp1_d);
	cutilCheckMsg("Kernel execution failed: [ nextstep3 ]");
	cufftSafeCall(cufftMyExecB(planz2d,(cufftCmplx *)temp1_d,(cufftReal_ *)v_d));

	nextstep4 <<< nBlocksC, nThreads >>>(Nx,Ny,psihat_d,kx_d,temp1_d);
	cutilCheckMsg("Kernel execution failed: [ nextstep4 ]");
	cufftSafeCall(cufftMyExecB(planz2d,(cufftCmplx *)temp1_d,(cufftReal_ *)u_d));


	//!!!!!!!!!!!!!!!!!!!!!!!!
	//!initial nonlinear term!
	//!!!!!!!!!!!!!!!!!!!!!!!!'
	nonlin1 <<< nBlocksC, nThreads >>>(Nx,kx_d,omeghat_d,temp1_d);
	cutilCheckMsg("Kernel execution failed: [ nonlin1 ]");
    cufftSafeCall(cufftMyExecB(planz2d,(cufftCmplx *)temp1_d,(cufftReal_ *)temp2_d));
	nonlin2 <<< nBlocksR, nThreads >>>(u_d,temp2_d,nl_d);
	cutilCheckMsg("Kernel execution failed: [ nonlin2 ]");
    nonlin3 <<< nBlocksC, nThreads >>>(Ny,ky_d,omeghat_d,temp1_d);
	cutilCheckMsg("Kernel execution failed: [ nonlin3 ]");
	cufftSafeCall(cufftMyExecB(planz2d,(cufftCmplx *)temp1_d,(cufftReal_ *)temp2_d));
	nonlin4 <<< nBlocksR, nThreads >>>(Nx,Ny,nl_d,v_d,temp2_d);
	cutilCheckMsg("Kernel execution failed: [ nonlin4 ]");
    cutilSafeCall(hipMemcpy(omeg, nl_d, Nx * Ny * sizeof(real), hipMemcpyDeviceToHost));
    
    
	cufftSafeCall(cufftMyExecF(pland2z,(cufftReal_ *)nl_d,(cufftCmplx *)nlhat_d));
	//!!!!!!!!!!!!!!!!!!!!!

	printf("Got initial data, starting timestepping\n");	
	gettimeofday(&start_time, NULL);
	
	for(nn=1; nn<=nplots; nn++){
		for(n=1; n<=plotgap; n++){
			chg=1.0;
			copyCmplxArray <<< nBlocksC, nThreads >>>(nloldhat_d,nlhat_d);
			cutilCheckMsg("Kernel execution failed: [ copyCmplxArray ]");
			copyCmplxArray <<< nBlocksC, nThreads >>>(omegoldhat_d,omeghat_d);
			cutilCheckMsg("Kernel execution failed: [ copyCmplxArray ]");
			while(chg>tol){
				//!!!!!!!!!!!!!!!!!!!!!!
				//!{n,k} nonlinear term!
				//!!!!!!!!!!!!!!!!!!!!!!
				nonlin1 <<< nBlocksC, nThreads >>>(Ny,kx_d,omeghat_d,temp1_d);
				cutilCheckMsg("Kernel execution failed: [ nonlin1 ]");
				cufftSafeCall(cufftMyExecB(planz2d, (cufftCmplx *)temp1_d, (cufftReal_ *)temp2_d));
				nonlin2 <<< nBlocksR, nThreads >>>(u_d, temp2_d, nl_d);
				cutilCheckMsg("Kernel execution failed: [ nonlin2 ]");
				nonlin3 <<< nBlocksC, nThreads >>>(Ny, ky_d, omeghat_d, temp1_d);
				cutilCheckMsg("Kernel execution failed: [ nonlin3 ]");
				cufftSafeCall(cufftMyExecB(planz2d, (cufftCmplx *)temp1_d, (cufftReal_ *)temp2_d));
				nonlin4 <<< nBlocksR, nThreads >>>(Nx, Ny, nl_d, v_d, temp2_d);
				cutilCheckMsg("Kernel execution failed: [ nonlin4 ]");
 				cutilSafeCall(hipMemcpy(omeg, nl_d, Nx * Ny * sizeof(real), hipMemcpyDeviceToHost));
    			
				cufftSafeCall(cufftMyExecF(pland2z, (cufftReal_ *)nl_d, (cufftCmplx *)nlhat_d));
				//!!!!!!!!!!!!!!!!!!!!!
				nextstep1 <<< nBlocksC, nThreads >>>(Nx,Ny,dt,Re,kx_d,ky_d,omeghat_d,omegoldhat_d,nloldhat_d,nlhat_d);
				cutilCheckMsg("Kernel execution failed: [ nextstep1 ]");
				// Calculate streamfunction in fourier space, psihat
		 		nextstep2 <<< nBlocksC, nThreads >>> (Nx, Ny, kx_d, ky_d, omeghat_d, psihat_d);
				cutilCheckMsg("Kernel execution failed: [ nextstep2 ]");
 				// Calculate y velocity
				nextstep3 <<< nBlocksC, nThreads >>>(Nx,Ny,psihat_d,kx_d,temp1_d);
				cutilCheckMsg("Kernel execution failed: [ nextstep3 ]");
				cufftSafeCall(cufftMyExecB(planz2d,(cufftCmplx *)temp1_d,(cufftReal_ *)v_d));
				// Calculate x velocity
				nextstep4 <<< nBlocksC, nThreads >>>(Nx,Ny,psihat_d,kx_d,temp1_d);
				cutilCheckMsg("Kernel execution failed: [ nextstep4 ]");
				cufftSafeCall(cufftMyExecB(planz2d,(cufftCmplx *)temp1_d,(cufftReal_ *)u_d));

				cufftSafeCall(cufftMyExecB(planz2d, (cufftCmplx *)omeghat_d, (cufftReal_ *)omeg_d));
				checkConvergence1 <<< nBlocksR, nThreads >>>(Nx, Ny, omeg_d, omegcheck_d);
				cutilCheckMsg("Kernel execution failed: [ checkConvergence1 ]");
				cufftSafeCall(cufftMyExecF(pland2z, (cufftReal_ *)omegcheck_d, (cufftCmplx *)temp1_d));
				cutilSafeCall(hipMemcpy(omeg, temp1_d, sizeof(real), hipMemcpyDeviceToHost));
				chg=omeg[0];				
				copyRealArray <<< nBlocksR, nThreads >>>(omegcheck_d, omeg_d);
				cutilCheckMsg("Kernel execution failed: [ copyRealArray ]");	
			}
		}	
	}
	
	gettimeofday(&end_time, NULL);
	start_time.tv_sec = end_time.tv_sec - start_time.tv_sec;
	start_time.tv_usec = end_time.tv_usec - start_time.tv_usec;
	printf ("Timstepping took %lf seconds...\n", (real)(start_time.tv_sec) + (real)(start_time.tv_usec) / (real)1000000.);
	
	cutilSafeCall(hipMemcpy(omeg, omeg_d, Nx * Ny * sizeof(real), hipMemcpyDeviceToHost));

	// get exact omega
	for(j=0; j<Ny; j++) {
		 for(i=0;i<Nx;i++){
			omegexact[j*Ny+i]=4.0*pi*sin(2.0*pi*x[i])*sin(2.0*pi*y[j])*exp(-8.0*pi*pi*(real)nplots*(real)plotgap*dt/Re);
		}
	}
	
	max=0;
	for(i=0; i<Nx*Ny;i++) {
		omeg[i]=omeg[i]/(real)(Nx*Ny);
		chg=abs(omeg[i]-omegexact[i]);
		if(chg>=max) 
			max=chg;
	} 
	printf("Maximum error %lf ...\n", max);
	
	// turn of saving data for benchmarking
	//savedata (Nx,Ny,0,omeg);
	printf("Saved to disk\n");
	cutilSafeCall (hipFree ((void*)kx_d));
	cutilSafeCall (hipFree ((void*)ky_d));
	cutilSafeCall (hipFree ((void*)x_d));
	cutilSafeCall (hipFree ((void*)y_d));
	cutilSafeCall (hipFree ((void*)u_d));
	cutilSafeCall (hipFree ((void*)v_d));
	cutilSafeCall (hipFree ((void*)temp1_d));
	cutilSafeCall (hipFree ((void*)temp2_d));
	cutilSafeCall (hipFree ((void*)omeg_d));
	cutilSafeCall (hipFree ((void*)nl_d));
	cutilSafeCall (hipFree ((void*)omegcheck_d));
	cutilSafeCall (hipFree ((void*)omegoldhat_d));
	cutilSafeCall (hipFree ((void*)nloldhat_d));
	cutilSafeCall (hipFree ((void*)nlhat_d));
	cutilSafeCall (hipFree ((void*)psihat_d));
	printf("Deallocated GPU arrays \n");

	cufftSafeCall (hipfftDestroy (pland2z));
	cufftSafeCall (hipfftDestroy (planz2d));
	printf("Destroyed CUFFT plans \n");
	free (kx);
	free (ky);
	free (x);
	free (y);
	free (omeg);
	free (omegexact);
	printf ("Deallocated CPU arrays\n");
	printf("End Program \n");
	return 0;
}
