#include "hip/hip_runtime.h"
/*--------------------------------------------------------------------
*
*
* PURPOSE
*
* This program solves nonlinear sine-Gordon equation in 2 dimensions
* u_{tt}-u_{xx}-u_{yy}=-sin(u)
* using a second order implicit-explicit time stepping scheme.
*
* The boundary conditions are u(x=0,y)=u(2*Lx*\pi,y),
*       u(x,y=0)=u(x,y=2*Ly*\pi)
* 
* AUTHORS
*
* B. Cloutier, B.K. Muite, P. Rigge
* 4 June 2012
*
* .. Scalars ..
*  Nx                           = number of modes in x - power of 2 for FFT
*  Ny                           = number of modes in y - power of 2 for FFT
*  Nt                           = number of timesteps to take
*  plotgap                      = number of timesteps between plots
*  Lx                           = width of box in x direction
*  Ly                           = width of box in y direction
*  i                            = loop counter in x direction
*  j                            = loop counter in y direction
*  n                            = loop counter for timesteps 
*  nThreads						= Number of threads for GPU to use
*  nBlocksR						= number of blocks for GPU to use for real arrays
*  nBlocksC						= number of blocks for GPU to use for complex arrays
*  planfc                       = Forward 2d fft plan  (FFTW)
*  planbc                       = Backward 2d fft plan (FFTW)
*  planf                        = Forward 2d fft plan  (CUFFT)
*  planb                        = Backward 2d fft plan (CUFFT)
*  dt                           = timestep
*  xsize    					= size of real arrays in x direction
*  ysize    					= size of real arrays in y direction
*  gridsize 					= size of array 
*  start_time					= start time of computation
*  end_time						= end time of evaluation
*  en                           = total energy
*  es       	                = strain energy
*  ep     	                    = potential energy
*  ek	                        = kinetic energy
*
* .. Arrays on CPU ..
*
*  u                            = approximate solution
*  uold                         = approximate solution at previous timestep
*  temp1                        = extra space for energy computation
*  temp2                        = extra space for energy computation
*  kx                           = fourier frequencies in x direction (real format)
*  ky                           = fourier frequencies in y direction (real format)
*  kx_c                         = fourier frequencies in x direction (complex format)
*  ky_c                         = fourier frequencies in y direction (complex format)
*  x                            = x locations
*  y                            = y locations
*
* .. Arrays on GPU ..
*
*  u_d							= approximate solution
*  v_d							= Fourier transform of approximate solution
*  vold_d						= Fourier transform of approximate solution at previous timestep
*  nonlinhat_d					= Fourier transform of nonlinear term
*  kx_d                         = fourier frequencies in x direction (on GPU)
*  ky_d                         = fourier frequencies in y direction (on GPU)
*
* REFERENCES
*
* ACKNOWLEDGEMENTS
*
* ACCURACY
*		
* ERROR INDICATORS AND WARNINGS
*
* FURTHER COMMENTS
*
* Check that the initial iterate is consistent with the
* boundary conditions for the domain specified
*
*--------------------------------------------------------------------
*
* External routines required
*       getgrid.f90     -- Get initial grid of points
*       initialdata.f90 -- Get initial data
*       enercalc.f90    -- Subroutine to calculate the energy
*       savedata.f90    -- Save initial data
*
* External libraries required
*       Cuda FFT        -- http://developer.nvidia.com/cufft
*       FFTW3           -- Fastest Fourier Transform in the West
*                       (http://www.fftw.org/)
*       OpenMP
*/

#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <sys/time.h>

#include <hipfft/hipfft.h>
#include <cutil_inline.h>
#include <fftw3.h>


typedef double2 cmplx;
typedef double real;
typedef hipfftDoubleReal cufftReal_;
typedef hipfftDoubleComplex cufftCmplx;
#define cufftMyExecF hipfftExecD2Z
#define cufftMyExecB hipfftExecZ2D
#define CUFFT_TFORM_FORWARD  HIPFFT_D2Z
#define CUFFT_TFORM_BACKWARD HIPFFT_Z2D


#define IDX(i,j,Nx) (Nx*(j)+(i))


extern "C" {
extern int enercalc(int*, int*, fftw_plan*, fftw_plan*,
                    double*, double*, double*, double*,
                    double*, double**, double**, cmplx**,
                    cmplx**, double**, double**);
}

static __device__ inline cmplx cadd (cmplx a, cmplx b) {
  cmplx c;
  c.x = a.x + b.x;
  c.y = a.y + b.y;
  return c;
}

static __device__ inline cmplx csub (cmplx a, cmplx b) {
  cmplx c;
  c.x = a.x - b.x;
  c.y = a.y - b.y;
  return c;
}

static __device__ inline cmplx cscale (cmplx a, real b) {
  cmplx c;
  c.x = a.x * b;
  c.y = a.y * b;
  return c;
}

/* Compute nonlinear term */
static __global__ void nonlinterm(real *u_d) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  u_d[idx] = -1.0*sin(u_d[idx]);
}

/* Scale data by 1/(Nx*Ny) after ifft */
static __global__ void scaledata(int Nx, int Ny, real *u) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  real s = (real)1. / ((real)Nx * (real)Ny);
  u[idx] = s * u[idx];
}

/* Compute next timestep in Fourier space */
static __global__ void nextstep(int Nx, int Ny, real dt,
								real *kx_d, real *ky_d,
                                cmplx *v_d, cmplx *vold_d,
                                cmplx *nonlinhat_d) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int i, j;
  i = idx / (Ny/2+1);
  j = idx % (Ny/2+1);
  /* Minus kx and minus ky because kx, ky multiplied by i=sqrt(-1) */
  real coef1 = (real)0.25 * (-kx_d[i]*kx_d[i] - ky_d[j]*ky_d[j]);
  real coef2 = (real)1./(dt*dt);
  real coef3 = (real)1.;
  real coef4 = (real)1. / (coef2 - coef1);
  coef1 = coef1 * coef4;
  coef2 = coef2 * coef4;
  coef3 = coef3 * coef4;
  cmplx term1 = cadd (cscale (v_d[idx], (real)2.), vold_d[idx]);
  cmplx term2 = csub (cscale (v_d[idx], (real)2.), vold_d[idx]);
  cmplx term3 = nonlinhat_d[idx];
  term1 = cscale (term1, coef1);
  term2 = cscale (term2, coef2);
  term3 = cscale (term3, coef3);
  vold_d[idx] = v_d[idx];
  v_d[idx] = cadd (cadd (term1, term2), term3);
  nonlinhat_d[idx] = cscale (v_d[idx], 1./ ((real)(Nx * Ny)));
}

/* Set u and uold to initial values */
void initialdata (int Nx, int Ny, real *x, real *y, real *u, real *uold) {
  int i, j;
  for (i=0; i<Nx; i++) {
    for (j=0; j<Ny; j++) {
      u[IDX(i,j,Nx)] = (real)0.5 * exp (-x[i]*x[i]-y[j]*y[j]);
      uold[IDX(i,j,Nx)] = u[IDX(i,j,Nx)];
    }
  }
}

/* Save u in binary format  */
void savedata (int Nx, int Ny, int nplot, real *u) {
  FILE *f = NULL;
  char nameconfig1[128];
  nameconfig1[0]='\0';
  sprintf (nameconfig1, "data/u%.10d.datbin", nplot);
  f = fopen (nameconfig1, "wb");
  fwrite (u, sizeof(real), Nx*Ny, f);
  fclose (f);
}

int main (int argc, char** argv) {
  hipfftHandle planf, planb;
  int Nx=1024;
  int Ny=1024;
  int Nt=500;
  int plotgap=500;
  double Lx=5.e0;
  double Ly=5.e0;
  double dt=1.e-3;
  real *kx, *ky, *kx_d, *ky_d, *x, *y, *kx_c, *ky_c;
  real *u, *u_d, *uold;
  double ek, es, ep, en;
  cmplx *vold_d, *v_d, *nonlinhat_d;;
  size_t xsize    = Nx * sizeof(real);
  size_t ysize    = Ny * sizeof(real);
  size_t gridsize = Nx * (Ny/2+1) * sizeof(cmplx);
  struct timeval start_time, end_time;
  int i, j, n;
  int nThreads, nBlocksR, nBlocksC;
  cmplx *temp1,*temp2;
  fftw_plan planfc, planbc;

  nThreads = (min(512, Nx));
  nBlocksR = (Nx * Ny / nThreads);
  nBlocksC = (Nx * (Ny/2+1) / nThreads);

  /* Print run information */
  printf ("Nx: %d\n", Nx);
  printf ("Ny: %d\n", Ny);
  printf ("Nt: %d\n", Nt);
  printf ("Lx: %lf\n", Lx);
  printf ("Ly: %lf\n", Ly);
  printf ("dt: %lf\n", dt);

  /* Allocate host arrays */
  kx  = (real*) malloc (xsize);
  ky  = (real*) malloc (ysize);
  kx_c  = (real*) malloc (xsize*2);
  ky_c  = (real*) malloc (ysize*2);
  x   = (real*) malloc (xsize);
  y   = (real*) malloc (ysize);
  u   = (real*) malloc (Nx * Ny * sizeof(real));
  uold= (real*) malloc (Nx * Ny * sizeof(real));
  temp1 = (cmplx*)malloc(Nx*Ny*sizeof(cmplx));
  temp2 = (cmplx*)malloc(Nx*Ny*sizeof(cmplx));

  /* Get cuda device */
  if (cutCheckCmdLineFlag (argc, (const char**)argv, "device")) {
    cutilDeviceInit(argc, argv);
  } else {
    hipSetDevice (cutGetMaxGflopsDeviceId());
  }

  /* Plan cuda FFTs */
  cufftSafeCall (
      hipfftPlan2d (&planf, Nx, Ny, CUFFT_TFORM_FORWARD));
  cufftSafeCall (
      hipfftPlan2d (&planb, Nx, Ny, CUFFT_TFORM_BACKWARD));

  /* Plan fftw FFTs */
  planfc=fftw_plan_dft_2d(Nx,Ny,(fftw_complex*)temp1,(fftw_complex*)temp2,FFTW_FORWARD,FFTW_ESTIMATE);
  planbc=fftw_plan_dft_2d(Nx,Ny,(fftw_complex*)temp2,(fftw_complex*)temp1,FFTW_BACKWARD,FFTW_ESTIMATE);
  printf ("Set up FFTs...\n");

  /* Allocate GPU arrays */
  cutilSafeCall (
      hipMalloc ((void**)&u_d, Nx * Ny * sizeof(cufftReal_)));
  cutilSafeCall (
      hipMalloc ((void**)&v_d,  gridsize));
  cutilSafeCall (
      hipMalloc ((void**)&vold_d,  gridsize));
  cutilSafeCall (
      hipMalloc ((void**)&nonlinhat_d,  gridsize));
  cutilSafeCall (
      hipMalloc ((void**)&kx_d, xsize));
  cutilSafeCall (
      hipMalloc ((void**)&ky_d, ysize));
  printf ("Allocated GPU arrays...\n");

  for (i=0; i<Nx/2; i++) {
    kx[i] = i/(real)Lx;
  }
  kx[Nx/2]=0.;
  for (i=0; i<Nx/2-1; i++) {
    kx[Nx/2+1+i] = -kx[Nx/2-i-1];
  }
  for (j=0; j<Ny/2; j++) {
    ky[j] = j/(real)Ly;
  }
  ky[Ny/2]=0.;
  for (j=0; j<Ny/2-1; j++) {
    ky[Ny/2+1+j] = -1*ky[Ny/2-j-1];
  }
  for (i=0; i<Nx; i++) {
    kx_c[2*i+0]=(real)0.;
    kx_c[2*i+1]=kx[i];
  }
  for (i=0; i<Ny; i++) {
    ky_c[2*i+0]=(real)0.;
    ky_c[2*i+1]=ky[i];
  }
  for (i=0; i<Nx; i++) {
    x[i] = (-1. + (2. * i)/(real)Nx) * M_PI * Lx;
  }
  for (j=0; j<Ny; j++) {
    y[j] = (-1. + (2. * j)/(real)Ny) * M_PI * Ly;
  }

  /* Set u, uold */
  initialdata (Nx,Ny,x,y,u,uold);
  /* savedata(Nx,Ny,0, u); */ /* disabled for benchmarking */

  cutilSafeCall (
      hipMemcpy (kx_d, kx, xsize, hipMemcpyHostToDevice));
  cutilSafeCall (
      hipMemcpy (ky_d, ky, ysize, hipMemcpyHostToDevice));
  cutilSafeCall (
      hipMemcpy (u_d, uold, Nx * Ny * sizeof(real), hipMemcpyHostToDevice));

  cufftSafeCall (
      cufftMyExecF (planf,
                    (cufftReal_ *)u_d,
                    (cufftCmplx *)vold_d));
  cutilSafeCall (
      hipMemcpy (u_d, u, Nx * Ny * sizeof(real), hipMemcpyHostToDevice));

  cufftSafeCall (
      cufftMyExecF (planf,
                    (cufftReal_ *)u_d,
                    (cufftCmplx *)v_d));
  cutilSafeCall (
      hipMemcpy (u_d, u, Nx * Ny * sizeof(real), hipMemcpyHostToDevice));

  printf ("Arrays initialized...\n");

  enercalc(&Nx,&Ny,&planfc,&planbc,&dt,&ek,&es,&ep,&en,
            &kx_c,&ky_c,&temp1,&temp2,&u,&uold);
  printf ("Initial Energy: Tot: %lf\tKin: %lf\tStr: %lf\tPot: %lf\n", en, ek, es, ep);

  gettimeofday(&start_time, NULL);
  for (n=0; n<Nt; n++) {
    if (((n+1)%plotgap) == 0 && 0) { /* turn of plotting for tests */
      cutilSafeCall (
          hipMemcpy (u, u_d, Nx * Ny * sizeof(real), hipMemcpyDeviceToHost));
      /* savedata(Nx,Ny,(n+1)/plotgap, u); */ /* disabled for benchmarking */
      printf ("%lf\n", (n+1)/plotgap);
    }

    nonlinterm <<< nBlocksR, nThreads >>> (u_d);
      cutilCheckMsg("Kernel execution failed: [ nonlinterm ]");

    cufftSafeCall (
        cufftMyExecF (planf,
                      (cufftReal_ *)u_d,
                      (cufftCmplx *)nonlinhat_d));

    nextstep <<< nBlocksC, nThreads >>>(Nx, Ny, dt, kx_d, ky_d, v_d, vold_d, nonlinhat_d);
    cutilCheckMsg("Kernel execution failed: [nextstep]");
    /* scaled v_d stored in nonlinhat_d */

    cufftSafeCall (
        cufftMyExecB (planb,
                      (cufftCmplx *)nonlinhat_d,
                      (cufftReal_ *)u_d));
  }
  gettimeofday(&end_time, NULL);
  start_time.tv_sec = end_time.tv_sec - start_time.tv_sec;
  start_time.tv_usec = end_time.tv_usec - start_time.tv_usec;

  printf ("Computation complete...\n");
  printf ("Computation took %lf seconds...\n", (real)(start_time.tv_sec) + (real)(start_time.tv_usec) / (real)1000000.);

  cufftSafeCall (
      cufftMyExecB (planb,
                    (cufftCmplx *)v_d,
                    (cufftReal_ *)u_d));
  scaledata <<< nBlocksR, nThreads >>> (Nx, Ny, u_d);
  cutilCheckMsg("Kernel execution failed: [scaledata]");
  cutilSafeCall (
      hipMemcpy (u, u_d, Nx * Ny * sizeof(real), hipMemcpyDeviceToHost));

  cufftSafeCall (
      cufftMyExecB (planb,
                   (cufftCmplx*)vold_d,
                    (cufftReal_*)u_d));
  scaledata <<< nBlocksR, nThreads >>> (Nx,Ny,u_d);
  cutilCheckMsg("Kernel execution failed: [scaledata]");
  cutilSafeCall (
      hipMemcpy (uold, u_d, Nx * Ny * sizeof(real), hipMemcpyDeviceToHost));

  enercalc(&Nx,&Ny,&planfc,&planbc,&dt,&ek,&es,&ep,&en,
            &kx_c,&ky_c,&temp1,&temp2,&u,&uold);
  printf ("Final Energy: Tot: %lf\tKin: %lf\tStr: %lf\tPot: %lf\n", en, ek, es, ep);

  cutilSafeCall (
      hipFree ((void*)kx_d));
  cutilSafeCall (
      hipFree ((void*)ky_d));
  cutilSafeCall (
      hipFree ((void*)u_d));
  cutilSafeCall (
      hipFree ((void*)v_d));
  cutilSafeCall (
      hipFree ((void*)vold_d));
  cutilSafeCall (
      hipFree ((void*)nonlinhat_d));
  cufftSafeCall (
      hipfftDestroy (planf));
  cufftSafeCall (
      hipfftDestroy (planb));
  fftw_destroy_plan(planfc);
  fftw_destroy_plan(planbc);
  free (kx);
  free (ky);
  free (kx_c);
  free (ky_c);
  free (x);
  free (y);
  free (u);
  free (uold);
  free (temp1);
  free (temp2);
  printf ("Cleaned up...\n");
  return 0;
}
