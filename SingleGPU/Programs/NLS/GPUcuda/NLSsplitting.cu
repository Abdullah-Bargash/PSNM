#include "hip/hip_runtime.h"
/*--------------------------------------------------------------------------
*
*
* PURPOSE
*
* A CUDA program to solve the nonlinear Schrodinger equation in 2 dimensions
* i*u_t+Es*|u|^2u+u_{xx}+u_{yy}=0
* using a second order time spectral splitting scheme. The program is based
* on a similar program to solve the Sine Gordon equation by P. Rigge
*
* The boundary conditions are u(x=0,y)=u(x=2*L*\pi,y) 
* and u(x,y=0)=u(x,y=2*L*\pi)
* The initial condition is u=exp(-x^2-y^2)
*
* AUTHORS
*
* B. Cloutier, B.K. Muite, P. Rigge
* 4 June 2012
*
* .. Scalars ..
*
*  Nx				= number of modes in x - power of 2 for FFT
*  Ny				= number of modes in y - power of 2 for FFT
*  dt				= timestep
*  Nt				= number of timesteps to take
*  plotgap			= number of timesteps between plots
*  Lx				= width of box in x direction
*  Ly				= width of box in y direction
*  ES				= +1 for focusing and -1 for defocusing
*  i				= loop counter in x direction
*  j				= loop counter in y direction
*  n				= loop counter for timesteps direction	
*  nThreads			= Number of threads for GPU to use
*  nBlocks			= number of blocks for GPU to use
*  plan				= fft plan
*  dt				= timestep
*  InMass			= initial mass
*  FiMass			= final mass
*  InEner			= initial energy
*  FiEner			= final energy
*  scalemodes		= scaled an array after performing inverse FFT
*  plan				= plan for fft
*  xsize    		= size of real arrays in x direction
*  ysize    		= size of real arrays in y direction
*  gridsize 		= size of array for complex data
*  complxsize 		= size of a complex data point 
*  start_time		= start time of computation
*  end_time			= end time of evaluation
*
* .. Arrays on CPU ..
*
*  u				= approximate solution
*  kx				= wave numbers in x direction
*  ky				= wave numbers in y direction
*
* .. Arrays on GPU ..
*
*  u_d				= approximate solution on device
*  v_d				= Fourier transform of approximate solution on device
*  temp1_d			= temporary array used to find mass and energy
*  temp2_d			= temporary array used to find mass and energy
*
* REFERENCES
*
* ACKNOWLEDGEMENTS
*
* ACCURACY
*		
* ERROR INDICATORS AND WARNINGS
*
* FURTHER COMMENTS
*
* Check that the initial condition is consistent with the 
* boundary conditions for the domain specified
*
* For consistency with Fortran programs, real is the same as double
* and complx is double2. The relevant complex arithmetic operations
* have been defined appropriately.
*
*--------------------------------------------------------------------
*
* External routines required
* 
* External libraries required
* cufft	 -- Cuda FFT library
*
*/

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>

#include <hipfft/hipfft.h>
#include <cutil_inline.h>

typedef double2 cmplx;
typedef double real;


static __device__ __host__ inline cmplx cmul (cmplx a, cmplx b) {
  cmplx c;
  c.x = a.x * b.x - a.y * b.y;
  c.y = a.x * b.y + a.y * b.x;
  return c;
}

static __device__ inline cmplx cscale (cmplx a, real b) {
  cmplx c;
  c.x = a.x * b;
  c.y = a.y * b;
  return c;
}

static __device__ inline cmplx cexp (cmplx a) {
  cmplx c;
  c.x = exp(a.x) * cos(a.y);
  c.y = exp(a.x) * sin(a.y);
  return c;
}

static __device__ inline real abssq (cmplx a) {
  real c;
  c = (a.x)*(a.x)+(a.y)*(a.y);
  return c;
}

static __global__ void potentialcal(cmplx *v_d, cmplx *u_d, real scalemodes, real Es) 
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  real temp;
  temp = abssq(u_d[idx]);
  v_d[idx].x = Es*temp*scalemodes*scalemodes;
  v_d[idx].y = 0;
}

static __global__ void uxencalc(cmplx *v_d, real *kx_d, cmplx *temp1_d,
                                real scalemodes, int Ny) 
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int i;
  i = idx / Ny;
  cmplx wavenum;
  wavenum.x = 0;
  wavenum.y = kx_d[i];
  cmplx temp;
  temp = cmul(wavenum,v_d[idx]);
  temp1_d[idx]=cscale(temp,scalemodes);
}

static __global__ void uyencalc(cmplx *v_d, real *ky_d, cmplx *temp1_d,
                                real scalemodes, int Ny) 
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int j;
  j = idx % Ny; 
  cmplx wavenum;
  wavenum.x = 0;
  wavenum.y = ky_d[j];
  cmplx temp;
  temp = cmul(wavenum,v_d[idx]);
  temp1_d[idx]=cscale(temp,scalemodes);
}

static __global__ void potencalc(cmplx *u_d, cmplx *temp1_d, real Es ) 
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  real temp;
  temp = -Es*(real)0.25*abssq(u_d[idx])*abssq(u_d[idx]);
  temp1_d[idx].x = temp;
  temp1_d[idx].y = 0;
}

static __global__ void abscalc(cmplx *u_d, cmplx *temp1_d ) 
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  real temp;
  temp = abssq(u_d[idx]);
  temp1_d[idx].x = temp;
  temp1_d[idx].y = 0;
}
  
static __global__ void realstep(cmplx *v_d, cmplx *u_d, real scalemodes, real dt) 
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  cmplx imag;
  imag.x = 0;
  imag.y = 1;
  cmplx temp1;
  temp1 = cscale(v_d[idx],dt);
  cmplx temp2;
  temp2 = cmul(imag, temp1);
  cmplx temp3;
  temp3 = cexp(temp2);
  cmplx temp4;
  temp4 = cmul(temp3,u_d[idx]);
  u_d[idx] = cscale(temp4,scalemodes);
}

static __global__ void fourierstep(real *kx_d, real *ky_d,
                   cmplx *v_d, int Nx, int Ny, real dt) 
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  cmplx imag;
  imag.x = 0;
  imag.y = 1;
  int i, j;
  i = idx / Ny;
  j = idx % Ny; 
  real wavenum;
  wavenum =  dt*(-kx_d[i]*kx_d[i] + -ky_d[j]*ky_d[j]);
  cmplx intfactor;
  intfactor = cexp(cscale(imag,wavenum));
  v_d[j*Nx+i]=cmul(intfactor,v_d[j*Nx+i]);
 }

static __global__ void fourierstephalf(real *kx_d, real *ky_d,
                   			cmplx *v_d, int Nx, int Ny, real dt) 
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  cmplx imag;
  imag.x = 0;
  imag.y = 1;
  int i, j;
  i = idx / Ny;
  j = idx % Ny; 
  real wavenum;
  wavenum =  (real)0.5*dt*(-kx_d[i]*kx_d[i] + -ky_d[j]*ky_d[j]);
  cmplx intfactor;
  intfactor = cexp(cscale(imag,wavenum));
  v_d[j*Nx+i]=cmul(intfactor,v_d[j*Nx+i]);
 }

static __global__ void scalefinal(cmplx *u_d, real scalemodes) 
{
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  u_d[idx] = cscale(u_d[idx],scalemodes);
}

void initialdata (int Nx, int Ny, real *x, real *y, cmplx *u) {
  int i, j;
  for (j=0; j<Ny; j++) {
    for (i=0; i<Nx; i++) {
      u[j*Nx+i].x = 0.5*exp(-1.0*(x[i]*x[i] + y[j]*y[j]));
    }
  }
}

void savedata (int Nx, int Ny, int nplot, cmplx *u) {
  FILE *f = NULL;
  char nameconfig1[128];
  nameconfig1[0]='\0';
  sprintf (nameconfig1, "data/u%.10d.datbin", nplot);
  f = fopen (nameconfig1, "wb");
  real *ureal;
  ureal = (real*)malloc (Nx * Ny * sizeof(real));
  int i;
  for (i=0; i<Nx*Ny; i++) {
    ureal[i] = u[i].x;
  }
  fwrite (ureal, sizeof(real), Nx*Ny, f);
  fclose (f);
}

int main (int argc, char** argv) {
  hipfftHandle plan;
  int Nx=128;
  int Ny=128;
  int Nt=20;
  int plotgap=1;
  int Lx=5.0;
  int Ly=5.0;
  real Es=1.0;
  real dt=1.e-5;
  real *kx, *ky, *kx_d, *ky_d, *x, *y;
  cmplx *u, *u_d, *v_d, *temp1_d, *temp2_d;
  size_t xsize    = Nx * sizeof(real);
  size_t ysize    = Ny * sizeof(real);
  size_t gridsize = Nx * Ny * sizeof(cmplx);
  size_t complxsize = sizeof(cmplx);
  struct timeval start_time, end_time;
  int nThreads, nBlocks;
  int i, j, n;
  real scalemodes;
  cmplx InMass, FiMass, InEner, FiEner;

  printf("Program starting\n");
  printf("Grid: %d X %d\n",Nx,Ny);
  printf("dt: %lf\n",dt);
  
  scalemodes=(real)1.0 / ( ( (real)Nx) * ( (real)Ny) );
  nThreads = 256;
  nBlocks  = Nx * Ny / nThreads;

  /* Allocate */
  kx = (real*) malloc (xsize);
  ky = (real*) malloc (ysize);
  x  = (real*) malloc (xsize);
  y  = (real*) malloc (ysize);
  u   = (cmplx*) malloc (gridsize);

  /* Plan FFTs */
  cufftSafeCall (
      hipfftPlan2d (&plan, Nx, Ny, HIPFFT_Z2Z));
  printf ("Set up FFTs...\n");

  cutilSafeCall (
      hipMalloc ((void**)&kx_d, xsize));
  cutilSafeCall (
      hipMalloc ((void**)&ky_d, ysize));
  cutilSafeCall (
      hipMalloc ((void**)&u_d,  gridsize));
  cutilSafeCall (
      hipMalloc ((void**)&v_d,  gridsize));
  cutilSafeCall (
      hipMalloc ((void**)&temp1_d,  gridsize));
  cutilSafeCall (
      hipMalloc ((void**)&temp2_d,  gridsize));
  printf ("Allocated GPU arrays...\n");

  /* Initialize arrays */
  for (i=0; i<Nx/2; i++) {
    kx[i] = i/(real)Lx;
  }
  kx[Nx/2]=0.;
  for (i=0; i<Nx/2-1; i++) {
    kx[Nx/2+1+i] = -kx[Nx/2-i-1];
  }
  for (i=0; i<Ny/2; i++) {
    ky[i] = i/(real)Ly;
  }
  ky[Ny/2]=0.;
  for (i=0; i<Ny/2-1; i++) {
    ky[Ny/2+1+i] = -ky[Ny/2-i-1];
  }
  for (i=0; i<Nx; i++) {
    x[i] = (-1. + (2. * i)/(real)Nx) * M_PI * Lx;
  }
  for (j=0; j<Ny; j++) {
    y[j] = (-1. + (2. * j)/(real)Ny) * M_PI * Ly;
  }
  printf ("Initialized arrays...\n");

  /* Initial data */
  initialdata (Nx, Ny, x, y, u);
  printf ("Got initial data...\n");
  
  savedata(Nx, Ny, 0, u);
  printf ("Saved initial data...\n");

  cutilSafeCall (
      hipMemcpy (kx_d, kx, xsize, hipMemcpyHostToDevice));
  cutilSafeCall (
      hipMemcpy (ky_d, ky, ysize, hipMemcpyHostToDevice));
  cutilSafeCall (
    hipMemcpy (u_d,  u, gridsize, hipMemcpyHostToDevice));
  printf ("Copied initial data to device...\n");
  cufftSafeCall (
      hipfftExecZ2Z (plan,
                    (hipfftDoubleComplex *)u_d,
                    (hipfftDoubleComplex *)v_d,
                    HIPFFT_FORWARD));

  printf ("Arrays initialized...\n");
  /* Calculate initial mass */
  abscalc <<< nBlocks, nThreads >>> (u_d, temp1_d);
  cutilCheckMsg("Kernel execution failed: [ abscalc ]");
    cufftSafeCall (
        hipfftExecZ2Z (plan,
                      (hipfftDoubleComplex *)temp1_d,
                      (hipfftDoubleComplex *)temp2_d,
                      HIPFFT_FORWARD));
  cutilSafeCall (
      hipMemcpy (u, temp2_d, complxsize, hipMemcpyDeviceToHost));
   InMass=u[0];
  /* Calculate initial energy */
  potencalc <<< nBlocks, nThreads >>> (u_d, temp1_d, Es);    
  cutilCheckMsg("Kernel execution failed: [ potencalc ]");
    cufftSafeCall (
        hipfftExecZ2Z (plan,
                      (hipfftDoubleComplex *)temp1_d,
                      (hipfftDoubleComplex *)temp2_d,
                      HIPFFT_FORWARD));
  cutilSafeCall (
      hipMemcpy (u, temp2_d, complxsize, hipMemcpyDeviceToHost));
  InEner=u[0];
  uxencalc <<< nBlocks, nThreads >>> (v_d, kx_d, temp1_d, scalemodes, Ny);    
  cutilCheckMsg("Kernel execution failed: [ uxencalc ]");
    cufftSafeCall (
        hipfftExecZ2Z (plan,
                      (hipfftDoubleComplex *)temp1_d,
                      (hipfftDoubleComplex *)temp2_d,
                      HIPFFT_BACKWARD));
  abscalc <<< nBlocks, nThreads >>> (temp2_d, temp1_d);
  cutilCheckMsg("Kernel execution failed: [ abscalc ]");
    cufftSafeCall (
        hipfftExecZ2Z (plan,
                      (hipfftDoubleComplex *)temp2_d,
                      (hipfftDoubleComplex *)temp1_d,
                      HIPFFT_FORWARD));
  cutilSafeCall (
      hipMemcpy (u, temp1_d, complxsize, hipMemcpyDeviceToHost));
   InEner.x=InEner.x+(real)0.5*u[0].x;
   InEner.y=InEner.y+(real)0.5*u[0].y;
  uyencalc <<< nBlocks, nThreads >>> (v_d, ky_d, temp1_d, scalemodes, Ny);    
  cutilCheckMsg("Kernel execution failed: [ uyencalc ]");
    cufftSafeCall (
        hipfftExecZ2Z (plan,
                      (hipfftDoubleComplex *)temp1_d,
                      (hipfftDoubleComplex *)temp2_d,
                      HIPFFT_BACKWARD));
  abscalc <<< nBlocks, nThreads >>> (temp2_d, temp1_d);
  cutilCheckMsg("Kernel execution failed: [ abscalc ]");
  cufftSafeCall (
        hipfftExecZ2Z (plan,
                      (hipfftDoubleComplex *)temp2_d,
                      (hipfftDoubleComplex *)temp1_d,
                      HIPFFT_FORWARD));
  cutilSafeCall (
      hipMemcpy (u, temp1_d, complxsize, hipMemcpyDeviceToHost));
   InEner.x=InEner.x+(real)0.5*u[0].x;
   InEner.y=InEner.y+(real)0.5*u[0].y;
  
                
  gettimeofday(&start_time, NULL);
  /* Do first half time step */	
  fourierstephalf <<< nBlocks, nThreads >>> (kx_d, ky_d, v_d, Nx, Ny, dt);
  cutilCheckMsg("Kernel execution failed: [ fourierstephalf ]");
   

  for (n=0; n<Nt; n++) {
    cufftSafeCall (
        hipfftExecZ2Z (plan,
                      (hipfftDoubleComplex *)v_d,
                      (hipfftDoubleComplex *)u_d,
                      HIPFFT_BACKWARD));
    potentialcal <<< nBlocks, nThreads >>>(v_d, u_d, scalemodes, Es);
    cutilCheckMsg("Kernel execution failed: [potentialcal]");
    realstep <<< nBlocks, nThreads >>>(v_d, u_d, scalemodes, dt);
    cutilCheckMsg("Kernel execution failed: [realstep]");
    cufftSafeCall (
        hipfftExecZ2Z (plan,
                      (hipfftDoubleComplex *)u_d,
                      (hipfftDoubleComplex *)v_d,
                      HIPFFT_FORWARD));
    fourierstep <<< nBlocks, nThreads >>> (kx_d, ky_d, v_d, Nx, Ny, dt);
    cutilCheckMsg("Kernel execution failed: [ fourierstep ]");
  }
  
  /* transform back final data and do another half step */
  cufftSafeCall (
        hipfftExecZ2Z (plan,
                      (hipfftDoubleComplex *)v_d,
                      (hipfftDoubleComplex *)u_d,
                      HIPFFT_BACKWARD));
  potentialcal <<< nBlocks, nThreads >>>(v_d, u_d, scalemodes, Es);
  cutilCheckMsg("Kernel execution failed: [potentialcal]");
  realstep <<< nBlocks, nThreads >>>(v_d, u_d, scalemodes, dt);
  cutilCheckMsg("Kernel execution failed: [realstep]");
  cufftSafeCall (
        hipfftExecZ2Z (plan,
                      (hipfftDoubleComplex *)u_d,
                      (hipfftDoubleComplex *)v_d,
                      HIPFFT_FORWARD));
  
  fourierstephalf <<< nBlocks, nThreads >>> (kx_d, ky_d, v_d, Nx, Ny, dt);
  cutilCheckMsg("Kernel execution failed: [ fourierstephalf ]");
  
  cufftSafeCall (
        hipfftExecZ2Z (plan,
                      (hipfftDoubleComplex *)v_d,
                      (hipfftDoubleComplex *)u_d,
                      HIPFFT_BACKWARD));
  scalefinal <<< nBlocks, nThreads >>>(u_d, scalemodes);
  
  gettimeofday(&end_time, NULL);
  start_time.tv_sec = end_time.tv_sec - start_time.tv_sec;
  start_time.tv_usec = end_time.tv_usec - start_time.tv_usec;

  printf ("Computation complete...\n");
  printf ("Computation took %lf seconds...\n",
  (real)(start_time.tv_sec) + (real)(start_time.tv_usec) / (real)1000000);
  cutilSafeCall (
          hipMemcpy ( u, u_d, gridsize, hipMemcpyDeviceToHost));
  savedata(Nx,Ny,1+n/plotgap, u); 
  /* Calculate final mass */
  abscalc <<< nBlocks, nThreads >>> (u_d, temp1_d);
  cutilCheckMsg("Kernel execution failed: [ abscalc ]");
    cufftSafeCall (
        hipfftExecZ2Z (plan,
                      (hipfftDoubleComplex *)temp1_d,
                      (hipfftDoubleComplex *)temp2_d,
                      HIPFFT_FORWARD));
  cutilSafeCall (
      hipMemcpy (u, temp2_d, complxsize, hipMemcpyDeviceToHost));
 FiMass=u[0];
   /* Calculate Final energy */
  potencalc <<< nBlocks, nThreads >>> (u_d, temp1_d, Es);    
  cutilCheckMsg("Kernel execution failed: [ potencalc ]");
    cufftSafeCall (
        hipfftExecZ2Z (plan,
                      (hipfftDoubleComplex *)temp1_d,
                      (hipfftDoubleComplex *)temp2_d,
                      HIPFFT_FORWARD));
  cutilSafeCall (
      hipMemcpy (u, temp2_d, complxsize, hipMemcpyDeviceToHost));
  FiEner=u[0];
  uxencalc <<< nBlocks, nThreads >>> (v_d, kx_d, temp1_d, scalemodes, Ny);    
  cutilCheckMsg("Kernel execution failed: [ uxencalc ]");
    cufftSafeCall (
        hipfftExecZ2Z (plan,
                      (hipfftDoubleComplex *)temp1_d,
                      (hipfftDoubleComplex *)temp2_d,
                      HIPFFT_BACKWARD));
  abscalc <<< nBlocks, nThreads >>> (temp2_d, temp1_d);
  cutilCheckMsg("Kernel execution failed: [ abscalc ]");
    cufftSafeCall (
        hipfftExecZ2Z (plan,
                      (hipfftDoubleComplex *)temp2_d,
                      (hipfftDoubleComplex *)temp1_d,
                      HIPFFT_FORWARD));
  cutilSafeCall (
      hipMemcpy (u, temp1_d, complxsize, hipMemcpyDeviceToHost));
   FiEner.x=FiEner.x+(real)0.5*u[0].x;
   FiEner.y=FiEner.y+(real)0.5*u[0].y;
  uyencalc <<< nBlocks, nThreads >>> (v_d, ky_d, temp1_d, scalemodes, Ny);    
  cutilCheckMsg("Kernel execution failed: [ uyencalc ]");
    cufftSafeCall (
        hipfftExecZ2Z (plan,
                      (hipfftDoubleComplex *)temp1_d,
                      (hipfftDoubleComplex *)temp2_d,
                      HIPFFT_BACKWARD));
  abscalc <<< nBlocks, nThreads >>> (temp2_d, temp1_d);
  cutilCheckMsg("Kernel execution failed: [ abscalc ]");
    cufftSafeCall (
        hipfftExecZ2Z (plan,
                      (hipfftDoubleComplex *)temp2_d,
                      (hipfftDoubleComplex *)temp1_d,
                      HIPFFT_FORWARD));
  cutilSafeCall (
      hipMemcpy (u, temp1_d, complxsize, hipMemcpyDeviceToHost));
   FiEner.x=FiEner.x+(real)0.5*u[0].x;
   FiEner.y=FiEner.y+(real)0.5*u[0].y;

 printf ("Initial mass %lf ...\n", InMass.x);
 printf ("Final mass %lf ...\n", FiMass.x);
 printf ("Initial energy %lf ...\n", InEner.x);
 printf ("Final energy %lf ...\n", FiEner.x);
 
 cutilSafeCall (
      hipFree ((void*)kx_d));
  cutilSafeCall (
      hipFree ((void*)ky_d));
  cutilSafeCall (
      hipFree ((void*)u_d));
  cutilSafeCall (
      hipFree ((void*)v_d));
  cutilSafeCall (
      hipFree ((void*)temp1_d));
  cutilSafeCall (
      hipFree ((void*)temp2_d));
  cufftSafeCall (
      hipfftDestroy (plan));
  free (kx);
  free (ky);
  free (x);
  free (y);
  free (u);
  printf ("Cleaned up...\n");
  return 0;
}
